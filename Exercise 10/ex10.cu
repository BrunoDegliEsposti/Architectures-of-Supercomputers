#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

double get_time(void) {
    struct timespec a;
    clock_gettime(CLOCK_MONOTONIC, &a);
    return (double)a.tv_sec + (double)a.tv_nsec * 1e-9;
}

__global__ void triad(double *A, double *B, double *C, double c, size_t nelems) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	for (; i < nelems; i += gridDim.x * blockDim.x) {
		A[i] = B[i] * c + C[i];
	}
}

int main(int argc, char **argv) {
	// CPU initialization
	size_t nbytes = 1024*1024*1024;
	size_t nelems = nbytes / sizeof(double);
	double *h_A = (double*)malloc(nbytes);
	double *h_B = (double*)malloc(nbytes);
	double *h_C = (double*)malloc(nbytes);
	double c = 0.0;
	for (size_t i = 0; i < nelems; i++) {
		h_B[i] = 1.0;
		h_C[i] = 1.0;
	}

	// GPU code
	double t0, t1, t2, t3;
	double *d_A, *d_B, *d_C;
	t0 = get_time();
	hipMalloc(&d_A, nbytes);
	hipMalloc(&d_B, nbytes);
	hipMalloc(&d_C, nbytes);
	hipMemcpy(d_B, h_B, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, nbytes, hipMemcpyHostToDevice);
	t1 = get_time();
	triad<<<nelems/1024, 1024>>>(d_A, d_B, d_C, c, nelems);
	hipDeviceSynchronize();
	t2 = get_time();
	hipMemcpy(h_A, d_A, nbytes, hipMemcpyDeviceToHost);
	t3 = get_time();

	// Error-checking
	hipError_t error = hipGetLastError();
	if(error != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	for (size_t i = 0; i < nelems; i++) {
		assert(abs(h_A[i] - h_B[i] * c - h_C[i]) < 1e-15);
	}
	
	// Statistics
	double mb1 = 3*1e-9*(double)nbytes/(t2-t1);
	double mb2 = 3*1e-9*(double)nbytes/(t3-t0);
	printf("Memory bandwidth (without memory transfers): %f GB/s\n", mb1);
	printf("Memory bandwidth (with memory transfers): %f GB/s\n", mb2);
	return 0;
}